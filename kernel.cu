#include "hip/hip_runtime.h"
//TODO kernel implementation
#define BLOCK_SIZE 32
#include <cmath>
__device__ float BUF[X*Y];
__device__ float gpresums[X];

__global__ void compute_cols(const float * __restrict__ din, float* dout, int x, int y, bool in_col){

    unsigned int col = blockIdx . x * blockDim.x + threadIdx.x;
    const float z = 1.73205080756887729f - 2.f; //sqrt(3)
    float *myCol = dout + col;

    // compute 'sum'
    float sum = (myCol[0*x] + powf(z, y)
                              * myCol[(y - 1)*x]) * 6 * (1.f + z) / z;
    int i = in_col ? col * x : col;
    sum += BUF[i] * 6;

    // iterate back and forth
    float cur;
    float last = sum * z / (1.f - powf(z, 2 * y));
    myCol[0] = last;
    for (int j = 1; j < y; ++j) {
        __syncthreads();
        cur = myCol[j*x] * 6 + z * last;
        myCol[j*x] = cur;
        last = cur;

    }
    __syncthreads();
    last = myCol[(y - 1)*x] * z / (z - 1.f);
    myCol[(y - 1)*x] = last;
    for (int j = y - 2; 0 <= j; --j) {
        __syncthreads();
        cur = z * (last - myCol[j*x]);
        myCol[j*x] = cur;
        last = cur;

    }
}

template<unsigned int blockSize>
__device__ inline void computesumrec(volatile float* sv, int tid){

    if ( tid < blockSize )
        sv [ tid ] += sv [ tid + blockSize / 2 ];
    __syncthreads();

    computesumrec<blockSize / 2>(sv, tid);
}

template<>
__device__ inline void computesumrec<1>(volatile float* sv, int tid) {}
#define SUMBLOCK 128


__global__ void compute_line_sum(const float * __restrict__ in, int x, bool first, int to_sum){
    __shared__ float sv [SUMBLOCK];
    if (!first)
        in = BUF;

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.y * x + blockDim.x * blockIdx.x + tid;

    float c = first ? gpresums[blockDim.x * blockIdx.x + tid] : 1;

    //copy to memory with gain if first
    float a =  blockDim.x * blockIdx.x + tid < to_sum ? in[i] : 0;
    sv [ tid ] = a * c;
    __syncthreads();

    computesumrec<SUMBLOCK/2>(sv, tid);
    if (tid == 0)
        BUF[blockIdx.y * x + blockIdx.x] = sv[0];
}


float arr[X];

#define TILE_DIM 32
#define BLOCK_ROWS 8
__global__ void transposeCoalesced(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

void solveGPU(float *in, float *out, int x, int y) {

//    hipMalloc(&buf, x * y * sizeof(float));
    dim3 a = dim3(x/32, y/32);
    dim3 b = dim3(32, 8);
    transposeCoalesced<<<a, b>>>(out, in);
    const float gain = 6.0f;
    const float z = 1.73205080756887729f - 2.f;
    float z1 = z;
    float z2 = powf(z, 2 * X - 2);
    float iz = 1.f / z;
    arr[0] = 0;
    arr[X-1] = 0;
    for (int j = 1; j < (X - 1); ++j) {
        arr[j] = (z2 + z1);
        z1 *= z;
        z2 *= iz;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(gpresums), arr, x * sizeof(float));

//    compute_presum<<<1, 1>>>(gpresums, x);

    bool first = true;
    for (int a = x; a > 0; a /= SUMBLOCK) {
        int aa = (a + SUMBLOCK - 1) / SUMBLOCK;
        dim3 dimgrid = dim3(aa, y);

        compute_line_sum<<<dimgrid, SUMBLOCK>>>(in, x, first, a);
//        printf("%d\n", aa);
        first = false;
    }
    compute_cols<<<x/BLOCK_SIZE, BLOCK_SIZE>>>(in, out, x, y, true);


    first = true;
    for (int a = x; a > 0; a /= SUMBLOCK) {
        int aa = (a + SUMBLOCK - 1) / SUMBLOCK;
        dim3 dimgrid = dim3(aa, y);

        compute_line_sum<<<dimgrid, SUMBLOCK>>>(out, x, first, a);
//        printf("%d\n", aa);
        first = false;
    }
    transposeCoalesced<<<a, b>>>(in, out);
    compute_cols<<<x/BLOCK_SIZE, BLOCK_SIZE>>>(in, in, x, y, true);
    hipMemcpy(out, in, x*y* sizeof(float), hipMemcpyDeviceToDevice);
}

