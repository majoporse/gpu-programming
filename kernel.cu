//TODO kernel implementation
#define BLOCK_SIZE 32

#include <hip/hip_runtime.h>
#include <cmath>

// copy input data
__global__ void copy_data(float* din, float* dout){
    unsigned int i = blockIdx . x* blockDim . x + threadIdx . x;
    dout[i] = din[i] * 6.0f;
}

__global__ void compute_lines(const float * __restrict__ din, float* __restrict__ dout, int x, int y){
    __ldg(din);
    unsigned int line = blockIdx . x * blockDim.x + threadIdx.x;
    float z = sqrtf(3.f) - 2.f;
    float z1;
    float* myLine = dout + (line * x);
    const float* myLineIn = din + (line * x);

    // compute 'sum'
    float sum = (myLineIn[0] + powf(z, x)
                             * myLineIn[x - 1]) * 6 * (1.f + z) / z;
    sum += dout[line * x] * 6;


    // iterate back and forth
    float last = sum * z / (1.f - powf(z, 2 * x));
    myLine[0] = last;
    float cur;
    for (int j = 1; j < x; ++j) {
        cur = myLineIn[j]* 6 + z * last;
        myLine[j] = cur;
        last = cur;
    }
    last = myLine[x - 1] * z / (z - 1.f);
    myLine[x - 1] =  last;
    for (int j = x - 2; 0 <= j; --j) {
        cur = z * (last - myLine[j]);
        myLine[j] = cur;
        last = cur;
    }
}

__global__ void compute_cols(const float * __restrict__ din, float* __restrict__ dout, int x, int y){
    __ldg(din);
    __ldg(dout);
    unsigned int col = blockIdx . x * blockDim.x + threadIdx.x;
    const float z = sqrtf(3.f) - 2.f;
//    float z1;
    float *myCol = dout + col;
    const float *myColIn = dout + col;

    // compute 'sum'
    float sum = (myColIn[0*x] + powf(z, y)
                              * myColIn[(y - 1)*x]) * 6 * (1.f + z) / z;

    sum += din[col] * 6;
//    if (col == 10){
//        printf("%f", sum);
//    }
    // iterate back and forth
    myCol[0*x] = sum * z / (1.f - powf(z, 2 * y));
    for (int j = 1; j < y; ++j) {
        myCol[j*x] = myColIn[j*x] * 6 + z * myCol[(j - 1)*x];
    }
    myCol[(y - 1)*x] *= z / (z - 1.f);
    for (int j = y - 2; 0 <= j; --j) {
        myCol[j*x] = z * (myCol[(j + 1)*x] - myCol[j*x]);
    }
}

__global__ void com(const float* presums, float* sums, const float * in, int x){
    __shared__ float cache[128];
    float sum = 0;
    int tid = threadIdx.x;
    int a = x / blockDim.x;
    int block_pos_x = blockIdx.x % a;
    int block_pos_y = blockIdx.x / a;
    cache[tid] = presums[block_pos_x * blockDim.x + tid];

    __syncthreads();
    for (int i = 0; i < 128; ++i){
        int cur = x * (block_pos_y * blockDim.x + tid) + block_pos_x * blockDim.x + i;
        sum += cache[i] * in[cur];
    }

    atomicAdd(&sums[tid + block_pos_y * blockDim.x], sum);
}

template<unsigned int blockSize>
__device__ inline void computesumrec(volatile float* sv, int tid){

    if ( tid < blockSize )
        sv [ tid ] += sv [ tid + blockSize / 2 ];
    __syncthreads();

    computesumrec<blockSize / 2>(sv, tid);
}

template<>
__device__ inline void computesumrec<1>(volatile float* sv, int tid) {}
#define SUMBLOCK 64

__global__ void compute_line_sum(float *in, float * __restrict__ presums, float* out, int x, bool first, int to_sum){
    __shared__ float sv [SUMBLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.y * x + blockDim.x * blockIdx.x + tid;

    float c = first ? presums[blockDim.x * blockIdx.x + tid] : 1;

    //copy to memory with gain if first
    float a =  blockDim.x * blockIdx.x + tid < to_sum ? in[i] : 0;
    sv [ tid ] = a * c;
    __syncthreads();

    computesumrec<SUMBLOCK/2>(sv, tid);
    if (tid == 0)
        out[blockIdx.y * x + blockIdx.x] = sv[0];
}

__global__ void compute_col_sum(float *in, float * __restrict__ presums, float* out, int x, bool first, int to_sum){
    __shared__ float sv [SUMBLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i =  blockIdx.x + (blockIdx.y * blockDim.x + tid) * x;

    float c = first ? presums[blockIdx.y * blockDim.x + tid] : 1;
    float a = blockIdx.y * blockDim.x + tid < to_sum ? in[i] : 0;
    //copy to memory with gain if first
    sv [ tid ] = a * c;
    __syncthreads();

    computesumrec<SUMBLOCK/2>(sv, tid);
    if (tid == 0)
        out[blockIdx.x + blockIdx.y * x] = sv[0];
}

__global__ void com2(const float* presums, float* sums, const float * in, int x){
    __shared__ float cache[128];
    float sum = 0;
    int tid = threadIdx.x;
    int a = x / blockDim.x;
    int block_pos_x = blockIdx.x % a;
    int block_pos_y = blockIdx.x / a;
    cache[tid] = presums[block_pos_y * blockDim.x + tid];

    __syncthreads();
    for (int i = 0; i < 128; ++i){
        int cur = x * block_pos_y * blockDim.x + block_pos_x * blockDim.x + tid + i * x;
        sum += cache[i] * in[cur];
    }

    atomicAdd(&sums[tid + block_pos_x * blockDim.x], sum);
}

__global__ void  compute_presum(float *  cpresums, int x){
    const float gain = 6.0f;
    const float z = sqrtf(3.f) - 2.f;
    float z1 = z;
    float z2 = powf(z, 2 * x - 2);
    float iz = 1.f / z;

    cpresums[0] = 0;
    cpresums[x-1] = 0;
    for (int j = 1; j < (x - 1); ++j) {
        cpresums[j] = (z2 + z1);
        z1 *= z;
        z2 *= iz;
    }
}

__device__ float* gpresums;
//__device__ float* sums;
//__device__ float* sums2;

void solveGPU(float *in, float *out, int x, int y) {

//    copy_data<<<(x*y)/256, 256>>>(in, out);


    hipMalloc(&gpresums, x * sizeof(float));
//    cudaMalloc(&sums, x * sizeof(float));
//    cudaMalloc(&sums2, x * sizeof(float));
//    cudaMemset(out, 0, x*y* sizeof(float));

    compute_presum<<<1,1>>>(gpresums, x);
    float * cur = in;
    bool first = true;
    for (int a = x; a > 0; a /= SUMBLOCK) {
        int aa = (a + SUMBLOCK - 1) / SUMBLOCK;
        dim3 dimgrid = dim3(aa, y);

        compute_line_sum<<<dimgrid, SUMBLOCK>>>(cur, gpresums, out, x, first, a);
//        printf("%d\n", aa);
        cur = out;
        first = false;
    }
//    com<<<(x*y)/(128 * 128), 128>>>(gpresums, sums, in, x);

    compute_lines<<<y/BLOCK_SIZE, BLOCK_SIZE>>>(in, out, x, y);

//    copy_data<<<(x*y)/256, 256>>>(in, out);
//    com2<<<(x*y)/(128 * 128), 128>>>(gpresums, sums2, out, x);
    cur = out;
    first = true;
    for (int a = x; a > 0; a /= SUMBLOCK) {
        int aa = (a + SUMBLOCK - 1) / SUMBLOCK;
        dim3 dimgrid = dim3(y, aa);

        compute_col_sum<<<dimgrid, SUMBLOCK>>>(cur, gpresums, in, x, first, a);
//        printf("%d\n", aa);
        cur = in;
        first = false;
    }
    compute_cols<<<x/BLOCK_SIZE, BLOCK_SIZE>>>(in, out, x, y);

}

